#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <vector>
#include <chrono>
using namespace std;

#define TILE_SIZE 32

__global__ void cuda(float* A, float* B, float* C, int N)
{
    __shared__ float a[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float b[TILE_SIZE][TILE_SIZE + 1];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int Col = bx * TILE_SIZE + tx;
    int Row = by * TILE_SIZE + ty;

    float sum = 0.0f;
    
    for(int i = 0; i < N / TILE_SIZE; i++)
    {
        a[ty][tx] = A[Row * N + (i * TILE_SIZE + tx)];
        b[ty][tx] = B[Col + (i * TILE_SIZE + ty) * N];
        __syncthreads();

        for(int k = 0; k < TILE_SIZE; k++)
            sum += a[ty][k] * b[k][tx];
        __syncthreads();
    }

    C[Col + N * Row ] = sum;
}

int main(int argc, char** argv) {
    const int N = 1024;
    vector<float> A(N*N);
    vector<float> B(N*N);
    vector<float> C(N*N, 0);

    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            A[N*i+j] = drand48();
            B[N*i+j] = drand48();
                                 }   
         }

    float *x, *y *z;
    hipMallocManaged(&x, N*N*sizeof(float));
    hipMallocManaged(&y, N*N*sizeof(float));
    hipMallocManaged(&z, N*N*sizeof(float));

    hipMemcpy(x,&A[0], N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y,&B[0], N*N*sizeof(float) hipMemcpyHostToDevice);
    auto tic = chrono::steady_clock::now();

    
    dim3 grid(8,8);
    dim3 block(TILE_SIZE, TILE_SIZE);
    cuda<<<grid, block>>>(N,x,y,z);
    hipDeviceSynchronize();

    auto toc = chrono::steady_clock::now();
    double comp_time = chrono::duration<double>(toc - tic).count();

    hipMemcpy(&C[0], z, sizeof(float) * N * N, hipMemcpyDeviceToHost);

    for (int i=0; i<N; i++)
        for (int j=0; j<N; j++)
            for (int k=0; k<N; k++)
                C[N*i+j] -= A[N*i+k] * B[N*k+j];

    double err = 0;
    for (int i=0; i<N; i++)
        for (int j=0; j<N; j++)
            err += fabs(C[i*N+j]);

    printf("N    : %d\n",N);
    printf("comp : %lf s\n", comp_time);
    printf("total: %lf s (%lf GFlops)\n", comp_time, 2.*N*N*N/comp_time/1e9);
    printf("error: %lf\n",err/N/N);
}
